#include <iostream>
#include <string>
#include <exception>
#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include <hip/hip_bf16.h>
#include <hiprand/hiprand.h>

#define CHECK_ERROR(error) \
    if (error != hipSuccess) { \
        cout << "ERROR:" << hipGetErrorString(error) << endl; \
        exit(-1); \
    }

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    exit(-1);}} while(0)

using std::string;
using std::cout;
using std::endl;

#ifdef __cplusplus
extern "C" {
    void burn(int gpu);
}
#endif

class BurnGPU {
private:
    int roundoff(int v, int d) {
        return (v + d - 1) / d * d;
    }

    hipError_t cuda_error {};  
    hipblasStatus_t cublas_status {};
    
    float* A = nullptr;
    float* B = nullptr;
    float* C = nullptr;
    hipblasComputeType_t cuCompType = HIPBLAS_COMPUTE_32F_FAST_16F;
    hipDataType cuDataType = HIP_R_32F;

    float alpha = 1.0;
    float beta = 0.0;

// matrix dims must agree with const int ld (see below)
// for the transpose and op states
#define SEED 10000

    const int Mm = SEED;
    const int Mn = SEED;
    const int Mk = SEED;
    const size_t As = SEED * SEED;
    const size_t Bs = SEED * SEED;
    const size_t Cs = SEED * SEED;

    hipblasLtMatmulDesc_t matmulDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL;
    hipblasLtMatrixLayout_t Bdesc = NULL;
    hipblasLtMatrixLayout_t Cdesc = NULL;
    hipblasLtHandle_t handle {};
    void* workspace;
    const size_t workspaceSize = 8192 * 8192 * 4;
    const hipblasOperation_t op = HIPBLAS_OP_N;
    const int ld = SEED;
    hipblasLtOrder_t order = HIPBLASLT_ORDER_COL;

public:
    BurnGPU(int gpu) {
        hipDeviceProp_t devprop {};
        CHECK_ERROR(hipSetDevice(gpu));
        CHECK_ERROR(hipGetDeviceProperties(&devprop, gpu));
        cout << "Found GPU " << gpu << " " << devprop.name << endl;

        CHECK_ERROR((hipMalloc((void**)&A, As)));
        CHECK_ERROR((hipMalloc((void**)&B, Bs)));
        CHECK_ERROR((hipMalloc((void**)&C, Cs)));
        CHECK_ERROR((hipMalloc((void**)&workspace, workspaceSize)));
    }

    void operator()() noexcept(false) {
        try {


            cublas_status = hipblasLtCreate(&handle);
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtCreate failed "
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatmulDescCreate(&matmulDesc,
                cuCompType,
                cuDataType);
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatmulDescCreate failed" 
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatmulDescSetAttribute(matmulDesc,
                HIPBLASLT_MATMUL_DESC_TRANSA, &op, sizeof(op));
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatmulDescSetAttribute A failed " 
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatmulDescSetAttribute(matmulDesc,
                HIPBLASLT_MATMUL_DESC_TRANSB, &op, sizeof(op));
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatmulDescSetAttribute B failed " 
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatmulDescSetAttribute(matmulDesc,
                CUBLASLT_MATMUL_DESC_TRANSC, &op, sizeof(op));
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatmulDescSetAttribute C failed "
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatrixLayoutCreate(&Adesc,
                    cuDataType,
                    Mm,
                    Mn,
                    ld);
            
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatrixLayoutCreate A failed " 
                        << cublas_status << endl;
                exit(-1);
            }
            cublas_status = hipblasLtMatrixLayoutSetAttribute(Adesc,
                    HIPBLASLT_MATRIX_LAYOUT_ORDER,
                    &order,
                    sizeof(hipblasLtOrder_t)
                    );
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatrixLayoutSetAttribute A failed "
                    << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatrixLayoutCreate(&Bdesc,
                    cuDataType,
                    Mn,
                    Mk,
                    ld);
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatrixLayoutCreate B failed " 
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatrixLayoutSetAttribute(Bdesc,
                HIPBLASLT_MATRIX_LAYOUT_ORDER,
                &order,
                sizeof(hipblasLtOrder_t)
                );
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatrixLayoutSetAttribute B failed "
                    << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatrixLayoutCreate(&Cdesc,
                                    cuDataType,
                    Mk,
                    Mm,
                    ld);
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatrixLayoutCreate C failed " 
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatrixLayoutSetAttribute(Cdesc,
                HIPBLASLT_MATRIX_LAYOUT_ORDER,
                &order,
                sizeof(hipblasLtOrder_t)
                );
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatrixLayoutSetAttribute C failed "
                    << cublas_status << endl;
                exit(-1);
            }

            while (true) {

                int seed = 0;
                hiprandGenerator_t prngGPU;
                CURAND_CALL(hiprandCreateGenerator(&prngGPU, HIPRAND_RNG_PSEUDO_MRG32K3A));
                CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(prngGPU, seed++));
                CURAND_CALL(hiprandGenerateUniform(prngGPU, (float *) A, As));
                CURAND_CALL(hiprandGenerateUniform(prngGPU, (float *) B, Bs));
                CURAND_CALL(hiprandGenerateUniform(prngGPU, (float *) C, Cs));
                cublas_status = hipblasLtMatmul(handle,
                        matmulDesc,
                    &alpha,
                    A,
                    Adesc,
                    B,
                    Bdesc,
                    &beta,
                    C,
                    Cdesc,
                    C,
                    Cdesc,
                    NULL,
                    workspace,
                    workspaceSize,
                    0);
                if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                    cout << "hipblasLtMatmul failed "
                        << cublas_status << endl;
                    exit(-1);
                }
            }
            CHECK_ERROR(hipDeviceSynchronize());

/*
            cout << "Done" << endl;
*/
                    cublas_status = hipblasLtDestroy(handle);
                    if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                            cout << "hipblasLtDestroy failed "
                                    << cublas_status << endl;
                            exit(-1);
                    }
        }
        catch (std::exception& e) {
            cout << "ERROR:" << e.what() << endl;
        }
    }

    ~BurnGPU() noexcept(false) {
        cublas_status = hipblasLtMatrixLayoutDestroy(Adesc);
        if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
            cout << "hipblasLtMatrixLayoutDestroy A failed "
                << cublas_status << endl;
            exit(-1);
        }

        cublas_status = hipblasLtMatrixLayoutDestroy(Bdesc);
        if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
            cout << "hipblasLtMatrixLayoutDestroy B failed "
                << cublas_status << endl;
            exit(-1);
        }

        cublas_status = hipblasLtMatrixLayoutDestroy(Cdesc);
        if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
            cout << "hipblasLtMatrixLayoutDestroy C failed "
                << cublas_status << endl;
            exit(-1);
        }

        cublas_status = hipblasLtMatmulDescDestroy(matmulDesc);
        if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
            cout << "hipblasLtMatmulDescDestroy failed cublas_status "
                << cublas_status << endl;
            exit(-1);
        }

        CHECK_ERROR(hipFree(workspace));
        CHECK_ERROR(hipFree(A));
        CHECK_ERROR(hipFree(B));
        CHECK_ERROR(hipFree(C));
    }

};

void burn(int gpu) {
    BurnGPU *burngpu = new BurnGPU(gpu);
    (*burngpu)();
}


#include <iostream>
#include <string>
#include <exception>
#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include <hip/hip_bf16.h>
#include <hiprand.h>
#include <unistd.h>
#include <time.h>
#include <sys/time.h>
#include <mpi.h>
#include <pthread.h>

#define M 1000000
#define K 1000

#define CHECK_ERROR(error) \
    if (error != hipSuccess) { \
        cout << "ERROR:" << hipGetErrorString(error) << endl; \
        exit(-1); \
    }

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    exit(-1);}} while(0)

using std::string;
using std::cout;
using std::endl;

#ifdef __cplusplus
extern "C" {
    void burn(int gpu, int cores, int low, double u_secs, double d_secs);
}
#endif

void *core_spin(void *target_ms) {
    timeval tod; 
    suseconds_t target_up_ms = *static_cast<suseconds_t *>(target_ms);
    gettimeofday(&tod, NULL);
    while(true) {
        if (target_up_ms <= tod.tv_sec * K + tod.tv_usec / K)  break;
        gettimeofday(&tod, NULL);
    }
    return(NULL);
}

#define CORE_SPIN() { \
    for (int thrix=0; thrix < cores; thrix++) {\
                ret = pthread_create(&tids[thrix], NULL, \
                        core_spin, (suseconds_t *) &target_up_ms);\
                if (ret != 0) {\
                    perror("burn pthread_create");\
                    exit(0);\
                } \
            }\
    }

class BurnGPU {
private:
    int roundoff(int v, int d) {
        return (v + d - 1) / d * d;
    }

    hipError_t cuda_error {};  
    hipblasStatus_t cublas_status {};
    
    float* A_up = nullptr;
    float* B_up = nullptr;
    float* C_up = nullptr;
    float* A_dn = nullptr;
    float* B_dn = nullptr;
    float* C_dn = nullptr;
    hipblasComputeType_t cuCompType = HIPBLAS_COMPUTE_32F_FAST_16F;
    hipDataType cuDataType = HIP_R_32F;

    float alpha = 1.0;
    float beta = 0.0;

    int cores = 0;
    int low = 0;

// matrix dims must agree with const int ld (see below)
// for the transpose and op states
#define SEED_UP 10000
#define SEED_DN_LOW 100
#define SEED_DN_HOT 1000

    const int Mm_up = SEED_UP;
    const int Mn_up = SEED_UP;
    const int Mk_up = SEED_UP;
    const size_t As_up = SEED_UP * SEED_UP;
    const size_t Bs_up = SEED_UP * SEED_UP;
    const size_t Cs_up = SEED_UP * SEED_UP;
    const int Mm_dn = (low) ?  SEED_DN_LOW : SEED_DN_HOT;
    const int Mn_dn = (low) ?  SEED_DN_LOW : SEED_DN_HOT;
    const int Mk_dn = (low) ?  SEED_DN_LOW : SEED_DN_HOT;
    const size_t As_dn = (low) ?  SEED_DN_LOW : SEED_DN_HOT * (low) ?  SEED_DN_LOW : SEED_DN_HOT;
    const size_t Bs_dn = (low) ?  SEED_DN_LOW : SEED_DN_HOT * (low) ?  SEED_DN_LOW : SEED_DN_HOT;
    const size_t Cs_dn = (low) ?  SEED_DN_LOW : SEED_DN_HOT * (low) ?  SEED_DN_LOW : SEED_DN_HOT;

    hipblasLtMatmulDesc_t matmulDesc_up = NULL;
    hipblasLtMatrixLayout_t Adesc_up = NULL;
    hipblasLtMatrixLayout_t Bdesc_up = NULL;
    hipblasLtMatrixLayout_t Cdesc_up = NULL;
    hipblasLtMatmulDesc_t matmulDesc_dn = NULL;
    hipblasLtMatrixLayout_t Adesc_dn = NULL;
    hipblasLtMatrixLayout_t Bdesc_dn = NULL;
    hipblasLtMatrixLayout_t Cdesc_dn = NULL;
    hipblasLtHandle_t handle_up {};
    hipblasLtHandle_t handle_dn {};
    void* workspace;
    const size_t workspaceSize = 8192 * 8192 * 4;
    const hipblasOperation_t op = HIPBLAS_OP_N;
    const int ld_up = SEED_UP;
    const int ld_dn = (low) ?  SEED_DN_LOW : SEED_DN_HOT;
    hipblasLtOrder_t order = HIPBLASLT_ORDER_COL;
    // for the square wave
    double up_seconds;
    double dn_seconds;
    int gpuid = -1;
    pthread_t* tids = 0;

public:
    BurnGPU(int gpu, int cores, int low, double u_secs, double d_secs) : cores(cores) {
        hipDeviceProp_t devprop {};
        CHECK_ERROR(hipSetDevice(gpu));
        CHECK_ERROR(hipGetDeviceProperties(&devprop, gpu));
        cout << "Found GPU " << gpu << " " << devprop.name << endl;
        cout << "Spinning " << cores << " CPU cores per GPU " << endl;

        if (cores) {
            tids = (pthread_t *) malloc(sizeof(pthread_t) * cores);
            if (! tids) {
                cout << "Failed to allocate memory for " << cores << " pthread_t " << endl;
                exit(-1);
            }
        }

	up_seconds = u_secs;
	dn_seconds = d_secs;
        gpuid = gpu;

        CHECK_ERROR((hipMalloc((void**)&A_up, As_up)));
        CHECK_ERROR((hipMalloc((void**)&B_up, Bs_up)));
        CHECK_ERROR((hipMalloc((void**)&C_up, Cs_up)));
        CHECK_ERROR((hipMalloc((void**)&A_dn, As_dn)));
        CHECK_ERROR((hipMalloc((void**)&B_dn, Bs_dn)));
        CHECK_ERROR((hipMalloc((void**)&C_dn, Cs_dn)));
        CHECK_ERROR((hipMalloc((void**)&workspace, workspaceSize)));
    }

    void operator()() noexcept(false) {
        try {

            cublas_status = hipblasLtCreate(&handle_up);
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtCreate failed "
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtCreate(&handle_dn);
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtCreate failed "
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatmulDescCreate(&matmulDesc_up,
                cuCompType,
                cuDataType);
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatmulDescCreate failed" 
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatmulDescCreate(&matmulDesc_dn,
                cuCompType,
                cuDataType);
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatmulDescCreate failed" 
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatmulDescSetAttribute(matmulDesc_up,
                HIPBLASLT_MATMUL_DESC_TRANSA, &op, sizeof(op));
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatmulDescSetAttribute A failed " 
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatmulDescSetAttribute(matmulDesc_dn,
                HIPBLASLT_MATMUL_DESC_TRANSA, &op, sizeof(op));
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatmulDescSetAttribute A failed " 
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatmulDescSetAttribute(matmulDesc_up,
                HIPBLASLT_MATMUL_DESC_TRANSB, &op, sizeof(op));
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatmulDescSetAttribute B failed " 
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatmulDescSetAttribute(matmulDesc_dn,
                HIPBLASLT_MATMUL_DESC_TRANSB, &op, sizeof(op));
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatmulDescSetAttribute B failed " 
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatmulDescSetAttribute(matmulDesc_up,
                CUBLASLT_MATMUL_DESC_TRANSC, &op, sizeof(op));
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatmulDescSetAttribute C failed "
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatmulDescSetAttribute(matmulDesc_dn,
                CUBLASLT_MATMUL_DESC_TRANSC, &op, sizeof(op));
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatmulDescSetAttribute C failed "
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatrixLayoutCreate(&Adesc_up,
                    cuDataType,
                    Mm_up,
                    Mn_up,
                    ld_up);
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatrixLayoutCreate A failed " 
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatrixLayoutCreate(&Adesc_dn,
                    cuDataType,
                    Mm_dn,
                    Mn_dn,
                    ld_dn);
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatrixLayoutCreate A failed " 
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatrixLayoutCreate(&Adesc_dn,
                    cuDataType,
                    Mm_dn,
                    Mn_dn,
                    ld_dn);
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatrixLayoutCreate A failed " 
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatrixLayoutSetAttribute(Adesc_up,
                    HIPBLASLT_MATRIX_LAYOUT_ORDER,
                    &order,
                    sizeof(hipblasLtOrder_t)
                    );
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatrixLayoutSetAttribute A failed "
                    << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatrixLayoutSetAttribute(Adesc_dn,
                    HIPBLASLT_MATRIX_LAYOUT_ORDER,
                    &order,
                    sizeof(hipblasLtOrder_t)
                    );
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatrixLayoutSetAttribute A failed "
                    << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatrixLayoutCreate(&Bdesc_up,
                    cuDataType,
                    Mn_up,
                    Mk_up,
                    ld_up);
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatrixLayoutCreate B failed " 
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatrixLayoutCreate(&Bdesc_dn,
                    cuDataType,
                    Mn_dn,
                    Mk_dn,
                    ld_dn);
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatrixLayoutCreate B failed " 
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatrixLayoutSetAttribute(Bdesc_up,
                HIPBLASLT_MATRIX_LAYOUT_ORDER,
                &order,
                sizeof(hipblasLtOrder_t)
                );
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatrixLayoutSetAttribute B failed "
                    << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatrixLayoutSetAttribute(Bdesc_dn,
                HIPBLASLT_MATRIX_LAYOUT_ORDER,
                &order,
                sizeof(hipblasLtOrder_t)
                );
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatrixLayoutSetAttribute B failed "
                    << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatrixLayoutCreate(&Cdesc_up,
                    cuDataType,
                    Mk_up,
                    Mm_up,
                    ld_up);
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatrixLayoutCreate C failed " 
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatrixLayoutCreate(&Cdesc_dn,
                    cuDataType,
                    Mk_dn,
                    Mm_dn,
                    ld_dn);
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatrixLayoutCreate C failed " 
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatrixLayoutSetAttribute(Cdesc_up,
                HIPBLASLT_MATRIX_LAYOUT_ORDER,
                &order,
                sizeof(hipblasLtOrder_t)
                );
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatrixLayoutSetAttribute C failed "
                    << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatrixLayoutSetAttribute(Cdesc_dn,
                HIPBLASLT_MATRIX_LAYOUT_ORDER,
                &order,
                sizeof(hipblasLtOrder_t)
                );
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatrixLayoutSetAttribute C failed "
                    << cublas_status << endl;
                exit(-1);
            }

            hiprandGenerator_t prngGPU;
            CURAND_CALL(hiprandCreateGenerator(&prngGPU, HIPRAND_RNG_PSEUDO_MRG32K3A));
            CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(prngGPU, 777));
            CURAND_CALL(hiprandGenerateUniform(prngGPU, (float *) A_up, As_up));
            CURAND_CALL(hiprandGenerateUniform(prngGPU, (float *) B_up, Bs_up));
            CURAND_CALL(hiprandGenerateUniform(prngGPU, (float *) C_up, Cs_up));
            CURAND_CALL(hiprandGenerateUniform(prngGPU, (float *) A_dn, As_dn));
            CURAND_CALL(hiprandGenerateUniform(prngGPU, (float *) B_dn, Bs_dn));
            CURAND_CALL(hiprandGenerateUniform(prngGPU, (float *) C_dn, Cs_dn));

            timeval tod;
            gettimeofday(&tod, NULL);
            int iterations = 1;
            /* Add one to the target up second and usleep to start on a second boundary with
            ** the target ms. set to 0; this elimnates the ms. slop comming out of the MPI barrier
            */
            suseconds_t target_up_ms = (suseconds_t) (((double) tod.tv_sec + up_seconds + 1.0) * K);
            printf("GPU %2d arrival second %ld ms. %ld target_up_ms %ld\n", gpuid, tod.tv_sec,
                    tod.tv_usec / K, target_up_ms);
	    usleep(M - tod.tv_usec);
	    printf("GPU %2d %sEntering loop. Up: %3.3f seconds. Down: %3.3f seconds.\n", 
                    gpuid, (ctime(&tod.tv_sec)), up_seconds, dn_seconds);
            int ret = 0;
            CORE_SPIN();
            while (iterations++) {
                cublas_status = hipblasLtMatmul(handle_up,
                    matmulDesc_up,
                    &alpha,
                    A_up,
                    Adesc_up,
                    B_up,
                    Bdesc_up,
                    &beta,
                    C_up,
                    Cdesc_up,
                    C_up,
                    Cdesc_up,
                    NULL,
                    workspace,
                    workspaceSize,
                    0);
                if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                    cout << "hipblasLtMatmul UP failed "
                        << cublas_status << endl;
                    exit(-1);
                }
                CHECK_ERROR(hipDeviceSynchronize());
                gettimeofday(&tod, NULL);
		if (target_up_ms <= tod.tv_sec * K + tod.tv_usec / K) {
                    suseconds_t target_dn_ms =(suseconds_t) ((double) tod.tv_sec * 
                            (double) K + (double) tod.tv_usec / (double) K + dn_seconds * (double) K);
                    printf("GPU %2d up phase done at ms. %ld target_dn_ms %ld Iterations %-8d\n",
                            gpuid, tod.tv_sec * K + tod.tv_usec / K, target_dn_ms, iterations);
                    iterations = 1;
                    for (int i=0; i < cores; i++) {
                        pthread_join(tids[i], (void **)NULL);
                    }
                    while(iterations) {
                        cublas_status = hipblasLtMatmul(handle_dn,
                            matmulDesc_dn,
                            &alpha,
                            A_dn,
                            Adesc_dn,
                            B_dn,
                            Bdesc_dn,
                            &beta,
                            C_dn,
                            Cdesc_dn,
                            C_dn,
                            Cdesc_dn,
                            NULL,
                            workspace,
                            workspaceSize,
                            0);
                        if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                            cout << "hipblasLtMatmul DN failed "
                                << cublas_status << endl;
                            exit(-1);
                        }
                        CHECK_ERROR(hipDeviceSynchronize());
                        gettimeofday(&tod, NULL);
        		if (target_dn_ms <= tod.tv_sec * K + tod.tv_usec / K)  break;
                    }
		    int rtn = MPI_Barrier(MPI_COMM_WORLD);
                    gettimeofday(&tod, NULL);
                    target_up_ms = (suseconds_t) ((double) tod.tv_sec *
                            (double) K + (double) tod.tv_usec / (double) K + up_seconds * (double) K);
                    printf("GPU %2d dn phase done at ms. %ld target_up_ms %ld\n", 
                            gpuid, tod.tv_sec * K + tod.tv_usec / K, target_up_ms);
                    CORE_SPIN(); 

	        }
            }
            CHECK_ERROR(hipDeviceSynchronize());

            cublas_status = hipblasLtDestroy(handle_up);
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtDestroy failed "
                        << cublas_status << endl;
                exit(-1);
            }
        }
        catch (std::exception& e) {
            cout << "ERROR:" << e.what() << endl;
        }
    }

    ~BurnGPU() noexcept(false) {
        free(tids);

        cublas_status = hipblasLtMatrixLayoutDestroy(Adesc_up);
        if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
            cout << "hipblasLtMatrixLayoutDestroy A failed "
                << cublas_status << endl;
            exit(-1);
        }

        cublas_status = hipblasLtMatrixLayoutDestroy(Adesc_dn);
        if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
            cout << "hipblasLtMatrixLayoutDestroy A failed "
                << cublas_status << endl;
            exit(-1);
        }

        cublas_status = hipblasLtMatrixLayoutDestroy(Bdesc_up);
        if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
            cout << "hipblasLtMatrixLayoutDestroy B failed "
                << cublas_status << endl;
            exit(-1);
        }

        cublas_status = hipblasLtMatrixLayoutDestroy(Bdesc_dn);
        if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
            cout << "hipblasLtMatrixLayoutDestroy B failed "
                << cublas_status << endl;
            exit(-1);
        }

        cublas_status = hipblasLtMatrixLayoutDestroy(Cdesc_up);
        if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
            cout << "hipblasLtMatrixLayoutDestroy C failed "
                << cublas_status << endl;
            exit(-1);
        }

        cublas_status = hipblasLtMatrixLayoutDestroy(Cdesc_dn);
        if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
            cout << "hipblasLtMatrixLayoutDestroy C failed "
                << cublas_status << endl;
            exit(-1);
        }

        cublas_status = hipblasLtMatmulDescDestroy(matmulDesc_up);
        if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
            cout << "hipblasLtMatmulDescDestroy failed cublas_status "
                << cublas_status << endl;
            exit(-1);
        }

        cublas_status = hipblasLtMatmulDescDestroy(matmulDesc_dn);
        if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
            cout << "hipblasLtMatmulDescDestroy failed cublas_status "
                << cublas_status << endl;
            exit(-1);
        }

        CHECK_ERROR(hipFree(workspace));
        CHECK_ERROR(hipFree(A_up));
        CHECK_ERROR(hipFree(B_up));
        CHECK_ERROR(hipFree(C_up));
        CHECK_ERROR(hipFree(A_dn));
        CHECK_ERROR(hipFree(B_dn));
        CHECK_ERROR(hipFree(C_dn));
    }

};

void burn(int gpu, int cores, int low, double u_secs, double d_secs) {
    BurnGPU *burngpu = new BurnGPU(gpu, cores, low, u_secs, d_secs);
    (*burngpu)();
}



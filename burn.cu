#include <iostream>
#include <string>
#include <exception>
#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include <hip/hip_bf16.h>
#include <hiprand.h>
#include <unistd.h>
#include <time.h>
#include <sys/time.h>
#include <mpi.h>

#define M 1000000
#define K 1000

#define CHECK_ERROR(error) \
    if (error != hipSuccess) { \
        cout << "ERROR:" << hipGetErrorString(error) << endl; \
        exit(-1); \
    }

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    exit(-1);}} while(0)

using std::string;
using std::cout;
using std::endl;

#ifdef __cplusplus
extern "C" {
    void burn(int gpu, int u_secs, int d_secs);
}
#endif

class BurnGPU {
private:
    int roundoff(int v, int d) {
        return (v + d - 1) / d * d;
    }

    hipError_t cuda_error {};  
    hipblasStatus_t cublas_status {};
    
    float* A = nullptr;
    float* B = nullptr;
    float* C = nullptr;
    hipblasComputeType_t cuCompType = HIPBLAS_COMPUTE_32F_FAST_16F;
    hipDataType cuDataType = HIP_R_32F;

    float alpha = 1.0;
    float beta = 0.0;

// matrix dims must agree with const int ld (see below)
// for the transpose and op states
#define SEED 10000

    const int Mm = SEED;
    const int Mn = SEED;
    const int Mk = SEED;
    const size_t As = SEED * SEED;
    const size_t Bs = SEED * SEED;
    const size_t Cs = SEED * SEED;

    hipblasLtMatmulDesc_t matmulDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL;
    hipblasLtMatrixLayout_t Bdesc = NULL;
    hipblasLtMatrixLayout_t Cdesc = NULL;
    hipblasLtHandle_t handle {};
    void* workspace;
    const size_t workspaceSize = 8192 * 8192 * 4;
    const hipblasOperation_t op = HIPBLAS_OP_N;
    const int ld = SEED;
    hipblasLtOrder_t order = HIPBLASLT_ORDER_COL;
    // for the square wave
    int up_seconds;
    int down_seconds;
    int gpuid = -1;

public:
    BurnGPU(int gpu, int u_secs, int d_secs) {
        hipDeviceProp_t devprop {};
        CHECK_ERROR(hipSetDevice(gpu));
        CHECK_ERROR(hipGetDeviceProperties(&devprop, gpu));
        cout << "Found GPU " << gpu << " " << devprop.name << endl;

	up_seconds = u_secs;
	down_seconds = d_secs;
        gpuid = gpu;

        CHECK_ERROR((hipMalloc((void**)&A, As)));
        CHECK_ERROR((hipMalloc((void**)&B, Bs)));
        CHECK_ERROR((hipMalloc((void**)&C, Cs)));
        CHECK_ERROR((hipMalloc((void**)&workspace, workspaceSize)));
    }

    void operator()() noexcept(false) {
        try {

            cublas_status = hipblasLtCreate(&handle);
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtCreate failed "
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatmulDescCreate(&matmulDesc,
                cuCompType,
                cuDataType);
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatmulDescCreate failed" 
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatmulDescSetAttribute(matmulDesc,
                HIPBLASLT_MATMUL_DESC_TRANSA, &op, sizeof(op));
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatmulDescSetAttribute A failed " 
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatmulDescSetAttribute(matmulDesc,
                HIPBLASLT_MATMUL_DESC_TRANSB, &op, sizeof(op));
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatmulDescSetAttribute B failed " 
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatmulDescSetAttribute(matmulDesc,
                CUBLASLT_MATMUL_DESC_TRANSC, &op, sizeof(op));
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatmulDescSetAttribute C failed "
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatrixLayoutCreate(&Adesc,
                    cuDataType,
                    Mm,
                    Mn,
                    ld);
            
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatrixLayoutCreate A failed " 
                        << cublas_status << endl;
                exit(-1);
            }
            cublas_status = hipblasLtMatrixLayoutSetAttribute(Adesc,
                    HIPBLASLT_MATRIX_LAYOUT_ORDER,
                    &order,
                    sizeof(hipblasLtOrder_t)
                    );
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatrixLayoutSetAttribute A failed "
                    << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatrixLayoutCreate(&Bdesc,
                    cuDataType,
                    Mn,
                    Mk,
                    ld);
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatrixLayoutCreate B failed " 
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatrixLayoutSetAttribute(Bdesc,
                HIPBLASLT_MATRIX_LAYOUT_ORDER,
                &order,
                sizeof(hipblasLtOrder_t)
                );
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatrixLayoutSetAttribute B failed "
                    << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatrixLayoutCreate(&Cdesc,
                                    cuDataType,
                    Mk,
                    Mm,
                    ld);
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatrixLayoutCreate C failed " 
                        << cublas_status << endl;
                exit(-1);
            }

            cublas_status = hipblasLtMatrixLayoutSetAttribute(Cdesc,
                HIPBLASLT_MATRIX_LAYOUT_ORDER,
                &order,
                sizeof(hipblasLtOrder_t)
                );
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtMatrixLayoutSetAttribute C failed "
                    << cublas_status << endl;
                exit(-1);
            }

            hiprandGenerator_t prngGPU;
            CURAND_CALL(hiprandCreateGenerator(&prngGPU, HIPRAND_RNG_PSEUDO_MRG32K3A));
            CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(prngGPU, 777));
            CURAND_CALL(hiprandGenerateUniform(prngGPU, (float *) A, As));
            CURAND_CALL(hiprandGenerateUniform(prngGPU, (float *) B, Bs));
            CURAND_CALL(hiprandGenerateUniform(prngGPU, (float *) C, Cs));

            timeval tod;
            gettimeofday(&tod, NULL);
            int iterations = 1;
	    int usleep_time = (down_seconds * M);
	    time_t t;
	    time(&t);
            /* Add one to the target up second and usleep to start on a secpnd boundary with
            ** the target ms. set to 0; this elimnates the ms. slop comming out of the MPI barrier
            */
            time_t target_up_second = t + (time_t) up_seconds + 1;
            suseconds_t target_up_ms = 0;
            printf("GPU %2d arrival second %ld ms. %3ld target_up_second %ld ms. %ld\n", gpuid, t,
                    tod.tv_usec / K, target_up_second, target_up_ms);
	    usleep(M - tod.tv_usec);
	    printf("GPU %2d %sEntering loop. Up: %d seconds. Down: %d seconds.\n", gpuid, (ctime(&t)), up_seconds, down_seconds);
            while (iterations++) {
                cublas_status = hipblasLtMatmul(handle,
                    matmulDesc,
                    &alpha,
                    A,
                    Adesc,
                    B,
                    Bdesc,
                    &beta,
                    C,
                    Cdesc,
                    C,
                    Cdesc,
                    NULL,
                    workspace,
                    workspaceSize,
                    0);
                if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                    cout << "hipblasLtMatmul failed "
                        << cublas_status << endl;
                    exit(-1);
                }
                CHECK_ERROR(hipDeviceSynchronize());
                time(&t);
                gettimeofday(&tod, NULL);
		if (target_up_second <= t && target_up_ms <= (tod.tv_usec / K)) {
                    printf("GPU %2d up phase done at second %ld ms. %3ld Iterations %-8d "
                           "target_up_second %ld target_up_ms %ld\n",
                            gpuid, t, tod.tv_usec / K, iterations, target_up_second, target_up_ms);
                    iterations = 1;
		    usleep(usleep_time);
		    int rtn = MPI_Barrier(MPI_COMM_WORLD);
                    time(&t);
                    gettimeofday(&tod, NULL);
                    target_up_second = t + up_seconds;
                    target_up_ms = tod.tv_usec / K;
                    printf("GPU %2d down phase done at second %ld ms. %3ld\n", 
                            gpuid, t, tod.tv_usec / K );
	        }
            }
            CHECK_ERROR(hipDeviceSynchronize());

            cublas_status = hipblasLtDestroy(handle);
            if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
                cout << "hipblasLtDestroy failed "
                        << cublas_status << endl;
                exit(-1);
            }
        }
        catch (std::exception& e) {
            cout << "ERROR:" << e.what() << endl;
        }
    }

    ~BurnGPU() noexcept(false) {
        cublas_status = hipblasLtMatrixLayoutDestroy(Adesc);
        if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
            cout << "hipblasLtMatrixLayoutDestroy A failed "
                << cublas_status << endl;
            exit(-1);
        }

        cublas_status = hipblasLtMatrixLayoutDestroy(Bdesc);
        if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
            cout << "hipblasLtMatrixLayoutDestroy B failed "
                << cublas_status << endl;
            exit(-1);
        }

        cublas_status = hipblasLtMatrixLayoutDestroy(Cdesc);
        if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
            cout << "hipblasLtMatrixLayoutDestroy C failed "
                << cublas_status << endl;
            exit(-1);
        }

        cublas_status = hipblasLtMatmulDescDestroy(matmulDesc);
        if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
            cout << "hipblasLtMatmulDescDestroy failed cublas_status "
                << cublas_status << endl;
            exit(-1);
        }

        CHECK_ERROR(hipFree(workspace));
        CHECK_ERROR(hipFree(A));
        CHECK_ERROR(hipFree(B));
        CHECK_ERROR(hipFree(C));
    }

};

void burn(int gpu, int u_secs, int d_secs) {
    // printf("BURN, gpu: %d, up seconds: %d, down_seconds: %d\n",gpu,u_secs,d_secs);
    BurnGPU *burngpu = new BurnGPU(gpu, u_secs, d_secs);
    (*burngpu)();
}


